
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cstdlib>
#include <iostream>
#define DLLEXPORT extern "C" __declspec(dllexport)

// C(rowsNum, colsNum) = A(rowsNum, width) * B(width, colsNum)
__global__ void matMul(float *A, float *B, float *C, int rowsNum, int width, int colsNum)
{
    unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < rowsNum && col < colsNum)
    {
        float value = 0;
        for (int k = 0; k < width; k++)
        {
            value += A[row * width + k] * B[k * colsNum + col];
        }

        C[row * colsNum + col] = value;
    }
}

DLLEXPORT void cudaMatMul(float *mat1, float *mat2, float *mat3, int mat1RowsNum, int mat1ColsNum, int mat2RowsNum, int mat2ColsNum)
{
    float *p_mat1, *p_mat2, *p_mat3;

    // Device memory allocation
    hipMalloc((void **)&p_mat1, mat1RowsNum * mat1ColsNum * sizeof(float));
    hipMalloc((void **)&p_mat2, mat2RowsNum * mat2ColsNum * sizeof(float));
    hipMalloc((void **)&p_mat3, mat1RowsNum * mat2ColsNum * sizeof(float));

    hipMemcpy(p_mat1, mat1, mat1RowsNum * mat1ColsNum * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(p_mat2, mat2, mat2RowsNum * mat2ColsNum * sizeof(float), hipMemcpyHostToDevice);

    // int devNo = 0;
    // cudaDeviceProp iProp;
    // cudaGetDeviceProperties(&iProp, devNo);

    // int BLOCK_SIZE = iProp.maxThreadsPerBlock;

    int BLOCK_SIZE = 16;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((mat2ColsNum + BLOCK_SIZE - 1) / BLOCK_SIZE, (mat1RowsNum + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matMul<<<gridDim, blockDim>>>(p_mat1, p_mat2, p_mat3, mat1RowsNum, mat1ColsNum, mat2ColsNum);

    hipMemcpy(mat3, p_mat3, mat1RowsNum * mat2ColsNum * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(p_mat1);
    hipFree(p_mat2);
    hipFree(p_mat3);
}
