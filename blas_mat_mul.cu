
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cstdlib>
#include <iostream>
#define DLLEXPORT extern "C" __declspec(dllexport)
using namespace std;


// C(rowsNum, colsNum) = A(rowsNum, width) * B(width, colsNum)
void blasMatMul(const float *A, const float *B, float *C, const int rowsNum, const int width, const int colsNum){
    const float alf = 1;
    const float bet = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
 
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, colsNum, rowsNum, width ,&alf ,B, colsNum, A, width, &bet, C, colsNum);

    hipblasDestroy(handle);
}



DLLEXPORT void cudaMatMul (float *mat1, float *mat2, float *mat3, int mat1RowsNum, int mat1ColsNum, int mat2RowsNum, int mat2ColsNum)
{   
  	float *p_mat1, *p_mat2, *p_mat3;

    //// device memory allocation
    hipMalloc((void**)&p_mat1, mat1RowsNum * mat1ColsNum * sizeof(float));
    hipMalloc((void**)&p_mat2, mat2RowsNum * mat2ColsNum * sizeof(float));
    hipMalloc((void**)&p_mat3, mat1RowsNum * mat2ColsNum * sizeof(float));

    hipMemcpy(p_mat1, mat1, mat1RowsNum * mat1ColsNum * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(p_mat2, mat2, mat2RowsNum * mat2ColsNum * sizeof(float), hipMemcpyHostToDevice);
   
    blasMatMul(p_mat1, p_mat2, p_mat3,  mat1RowsNum, mat1ColsNum, mat2ColsNum);

    hipMemcpy(mat3, p_mat3, mat1RowsNum * mat2ColsNum * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(p_mat1);
    hipFree(p_mat2);
    hipFree(p_mat3);
}





